#include "hip/hip_runtime.h"
/*
Matthew Golden
June 2017
*/

#ifndef __CUDA__EMISSION__INCLUDED__
#define __CUDA__EMISSION__INCLUDED__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include "emissionfunction.h" //Includes main.h ParameterReader.h Table.h

#define threadsPerBlock 512 //try optimizing this
//The number of blocks needed is calculated
#define debug 0	//1 for debugging, 0 otherwise

/*
This is a header file that defines the function calculate_dN_ptdptdphidy_GPU() for the EmissionFunctionArray

First, all needed information is arranged into arrays, then device copies are made.

The first  kernel performs the Cooper-Frye integral in parallel.
Each thread is assigned a cell of the freezeout surface.
The thread then loops over all particles and momenta.

After all threads have computed a set contributions to a certain momenta and mass, it sums over threads.
These partial sums are stored in an array dN_pTdpTdphidy[numofparticle * numofmomenta * numofblocks]
numofblocks is the number of blocks used to contain the freezeout surface

The next kernel reduction then does the secondary sum over the block dimension to yield a final complete spectrum
This is then copied back to the host and written to a file.
*/

__device__ void getbulkvisCoefficients(double Tdec, double* bulkvisCoefficients, double hbarC, int bulk_deltaf_kind)
{
   double Tdec_fm = Tdec / hbarC;  // [1/fm]
   double Tdec_fm_power[11];    // cache the polynomial power of Tdec_fm
   Tdec_fm_power[1] = Tdec_fm;
   for(int ipower = 2; ipower < 11; ipower++)
       Tdec_fm_power[ipower] = Tdec_fm_power[ipower-1] * Tdec_fm;
   /*if(bulk_deltaf_kind == 0)       // 14 moment expansion
   {
        // load from file
        bulkvisCoefficients[0] = bulkdf_coeff->interp(1, 2, Tdec_fm, 5)/pow(hbarC, 3);  //B0 [fm^3/GeV^3]
        bulkvisCoefficients[1] = bulkdf_coeff->interp(1, 3, Tdec_fm, 5)/pow(hbarC, 2);  // D0 [fm^3/GeV^2]
        bulkvisCoefficients[2] = bulkdf_coeff->interp(1, 4, Tdec_fm, 5)/pow(hbarC, 3);  // E0 [fm^3/GeV^3]
        // parameterization for mu = 0
        //bulkvisCoefficients[0] = exp(-15.04512474*Tdec_fm + 11.76194266)/pow(hbarC, 3); //B0[fm^3/GeV^3]
        //bulkvisCoefficients[1] = exp( -12.45699277*Tdec_fm + 11.4949293)/hbarC/hbarC;  // D0 [fm^3/GeV^2]
        //bulkvisCoefficients[2] = -exp(-14.45087586*Tdec_fm + 11.62716548)/pow(hbarC, 3);  // E0 [fm^3/GeV^3]
   }*/
   // use else if if previous code block is uncommented
   if(bulk_deltaf_kind == 1)  // relaxation type
   {
       // parameterization from JF
       // A Polynomial fit to each coefficient -- X is the temperature in fm^-1
       // Both fits are reliable between T=100 -- 180 MeV , do not trust it beyond
       bulkvisCoefficients[0] = (  642096.624265727
                                 - 8163329.49562861 * Tdec_fm_power[1]
                                 + 47162768.4292073 * Tdec_fm_power[2]
                                 - 162590040.002683 * Tdec_fm_power[3]
                                 + 369637951.096896 * Tdec_fm_power[4]
                                 - 578181331.809836 * Tdec_fm_power[5]
                                 + 629434830.225675 * Tdec_fm_power[6]
                                 - 470493661.096657 * Tdec_fm_power[7]
                                 + 230936465.421 * Tdec_fm_power[8]
                                 - 67175218.4629078 * Tdec_fm_power[9]
                                 + 8789472.32652964 * Tdec_fm_power[10]);

       bulkvisCoefficients[1] = (  1.18171174036192
                                 - 17.6740645873717 * Tdec_fm_power[1]
                                 + 136.298469057177 * Tdec_fm_power[2]
                                 - 635.999435106846 * Tdec_fm_power[3]
                                 + 1918.77100633321 * Tdec_fm_power[4]
                                 - 3836.32258307711 * Tdec_fm_power[5]
                                 + 5136.35746882372 * Tdec_fm_power[6]
                                 - 4566.22991441914 * Tdec_fm_power[7]
                                 + 2593.45375240886 * Tdec_fm_power[8]
                                 - 853.908199724349 * Tdec_fm_power[9]
                                 + 124.260460450113 * Tdec_fm_power[10]);
   }
   else if (bulk_deltaf_kind == 2)
   {
       // A Polynomial fit to each coefficient -- Tfm is the temperature in fm^-1
       // Both fits are reliable between T=100 -- 180 MeV , do not trust it beyond
       bulkvisCoefficients[0] = (
               21091365.1182649 - 290482229.281782 * Tdec_fm_power[1]
             + 1800423055.01882 * Tdec_fm_power[2] - 6608608560.99887 * Tdec_fm_power[3]
             + 15900800422.7138 * Tdec_fm_power[4] - 26194517161.8205 * Tdec_fm_power[5]
             + 29912485360.2916 * Tdec_fm_power[6] - 23375101221.2855 * Tdec_fm_power[7]
             + 11960898238.0134 * Tdec_fm_power[8] - 3618358144.18576 * Tdec_fm_power[9]
             + 491369134.205902 * Tdec_fm_power[10]);

       bulkvisCoefficients[1] = (
               4007863.29316896 - 55199395.3534188 * Tdec_fm_power[1]
             + 342115196.396492 * Tdec_fm_power[2] - 1255681487.77798 * Tdec_fm_power[3]
             + 3021026280.08401 * Tdec_fm_power[4] - 4976331606.85766 * Tdec_fm_power[5]
             + 5682163732.74188 * Tdec_fm_power[6] - 4439937810.57449 * Tdec_fm_power[7]
             + 2271692965.05568 * Tdec_fm_power[8] - 687164038.128814 * Tdec_fm_power[9]
             + 93308348.3137008 * Tdec_fm_power[10]);
   }
   else if (bulk_deltaf_kind == 3)
   {
       bulkvisCoefficients[0] = (
               160421664.93603 - 2212807124.97991 * Tdec_fm_power[1]
             + 13707913981.1425 * Tdec_fm_power[2] - 50204536518.1767 * Tdec_fm_power[3]
             + 120354649094.362 * Tdec_fm_power[4] - 197298426823.223 * Tdec_fm_power[5]
             + 223953760788.288 * Tdec_fm_power[6] - 173790947240.829 * Tdec_fm_power[7]
             + 88231322888.0423 * Tdec_fm_power[8] - 26461154892.6963 * Tdec_fm_power[9]
             + 3559805050.19592 * Tdec_fm_power[10]);
       bulkvisCoefficients[1] = (
               33369186.2536556 - 460293490.420478 * Tdec_fm_power[1]
             + 2851449676.09981 * Tdec_fm_power[2] - 10443297927.601 * Tdec_fm_power[3]
             + 25035517099.7809 * Tdec_fm_power[4] - 41040777943.4963 * Tdec_fm_power[5]
             + 5682163732.74188 * Tdec_fm_power[6] - 4439937810.57449 * Tdec_fm_power[7]
             + 2271692965.05568 * Tdec_fm_power[8] - 687164038.128814 * Tdec_fm_power[9]
             + 93308348.3137008 * Tdec_fm_power[10]);
   }
   else if (bulk_deltaf_kind == 3)
   {
       bulkvisCoefficients[0] = (
               160421664.93603 - 2212807124.97991 * Tdec_fm_power[1]
             + 13707913981.1425 * Tdec_fm_power[2] - 50204536518.1767 * Tdec_fm_power[3]
             + 120354649094.362 * Tdec_fm_power[4] - 197298426823.223 * Tdec_fm_power[5]
             + 223953760788.288 * Tdec_fm_power[6] - 173790947240.829 * Tdec_fm_power[7]
             + 88231322888.0423 * Tdec_fm_power[8] - 26461154892.6963 * Tdec_fm_power[9]
             + 3559805050.19592 * Tdec_fm_power[10]);
       bulkvisCoefficients[1] = (
               33369186.2536556 - 460293490.420478 * Tdec_fm_power[1]
             + 2851449676.09981 * Tdec_fm_power[2] - 10443297927.601 * Tdec_fm_power[3]
             + 25035517099.7809 * Tdec_fm_power[4] - 41040777943.4963 * Tdec_fm_power[5]
             + 46585225878.8723 * Tdec_fm_power[6] - 36150531001.3718 * Tdec_fm_power[7]
             + 18353035766.9323 * Tdec_fm_power[8] - 5504165325.05431 * Tdec_fm_power[9]
             + 740468257.784873 * Tdec_fm_power[10]);
   }
   else if (bulk_deltaf_kind == 4)
   {
       bulkvisCoefficients[0] = (
               1167272041.90731 - 16378866444.6842 * Tdec_fm_power[1]
             + 103037615761.617 * Tdec_fm_power[2] - 382670727905.111 * Tdec_fm_power[3]
             + 929111866739.436 * Tdec_fm_power[4] - 1540948583116.54 * Tdec_fm_power[5]
             + 1767975890298.1 * Tdec_fm_power[6] - 1385606389545 * Tdec_fm_power[7]
             + 709922576963.213 * Tdec_fm_power[8] - 214726945096.326 * Tdec_fm_power[9]
             + 29116298091.9219 * Tdec_fm_power[10]);
       bulkvisCoefficients[1] = (
               5103633637.7213 - 71612903872.8163 * Tdec_fm_power[1]
             + 450509014334.964 * Tdec_fm_power[2] - 1673143669281.46 * Tdec_fm_power[3]
             + 4062340452589.89 * Tdec_fm_power[4] - 6737468792456.4 * Tdec_fm_power[5]
             + 7730102407679.65 * Tdec_fm_power[6] - 6058276038129.83 * Tdec_fm_power[7]
             + 3103990764357.81 * Tdec_fm_power[8] - 938850005883.612 * Tdec_fm_power[9]
             + 127305171097.249 * Tdec_fm_power[10]);
   }
   return;
 }

__global__ void cooperFrye3D( long FO_length, int number_of_chosen_particles, int pT_tab_length, int phi_tab_length, int y_tab_length,
                              double* dN_pTdpTdphidy_d, double* pT_d, double* trig_d, double* y_d,
                              double* mass_d, double* sign_d, double* degen_d, int* baryon_d,
                              double *Tdec_d, double *Pdec_d, double *Edec_d, double *mu_d, double *tau_d, double *eta_d,
                              double *utau_d, double *ux_d, double *uy_d, double *ueta_d,
                              double *datau_d, double *dax_d, double *day_d, double *daeta_d,
                              double *pi00_d, double *pi01_d, double *pi02_d, double *pi11_d, double *pi12_d, double *pi22_d, double *pi33_d,
                              double *muB_d, double *bulkPi_d,
                              double hbarC, int bulk_deltaf_kind, int INCLUDE_DELTAF, int INCLUDE_BULKDELTAF, int F0_IS_NOT_SMALL)
{
  //This array is a shared array that will contain the integration contributions from each cell.
  __shared__ double temp[threadsPerBlock];

  //Assign a global index and a local index
  int idx_glb = threadIdx.x + blockDim.x * blockIdx.x;
  int icell = threadIdx.x;
  __syncthreads();

	//Declare things that do not depend on momentum outside of loop
  double bulkvisCoefficients[3] = {0.,0.,0.};
  if (INCLUDE_BULKDELTAF == 1)
  {
    if (bulk_deltaf_kind != 0) bulkPi_d[icell] = bulkPi_d[icell] / hbarC;   // unit in fm^-4
    getbulkvisCoefficients(Tdec_d[icell], bulkvisCoefficients, hbarC, bulk_deltaf_kind);
  }
	double deltaf_prefactor = 1.0/( 2.0 * Tdec_d[icell] * Tdec_d[icell] * (Edec_d[icell] + Pdec_d[icell]) );

  for (long imm = 0; imm < number_of_chosen_particles * pT_tab_length * phi_tab_length * y_tab_length; imm++) //this index runs over all particle species and momenta
  {
    temp[icell] = 0.0;
    if (icell < FO_length) //this index corresponds to the freezeout cell
    {
      //imm = ipT + (iphip * (pT_tab_length)) + (iy * (pT_tab_length * phi_tab_length)) + (ipart * (pT_tab_length * phi_tab_length * y_tab_length))
      int ipart       = imm / (pT_tab_length * phi_tab_length * y_tab_length);
      int iy          = (imm - (ipart * pT_tab_length * phi_tab_length * y_tab_length) ) / (pT_tab_length * phi_tab_length);
      int iphip       = (imm - (ipart * pT_tab_length * phi_tab_length * y_tab_length) - (iy * pT_tab_length * phi_tab_length) ) / pT_tab_length;
      int ipT         = imm - ( (ipart * (pT_tab_length * phi_tab_length * y_tab_length)) + (iy * (pT_tab_length * phi_tab_length)) + (iphip * (pT_tab_length)) );
      double px       = pT_d[ipT] * trig_d[ipT + phi_tab_length];
      double py       = pT_d[ipT] * trig_d[ipT];
      double mT       = sqrt(mass_d[ipart] * mass_d[ipart] + pT_d[ipT] * pT_d[ipT]);
      double y        = y_d[iy];
      double ptau     = mT * cosh(y - eta_d[icell]); //contravariant
      double peta     = (-1.0 / tau_d[icell]) * mT * sinh(y - eta[icell]); //contravariant

      double pdotu = ptau * utau[icell] - px * ux[icell] - py * uy[icell] - (tau[icell] * tau[icell]) * peta * ueta[icell]; //watch factors of tau from metric! is ueta read in as contravariant?
      double expon = (pdotu - mu_d[icell] - baryon_d[ipart] * muB_d[icell]) / Tdec_d[icell];
      //thermal equilibrium distributions
      double f0 = 1./(exp(expon) + sign_d[ipart]);
      double pdotdsigma = ptau * datau[icell] + px * dax[icell] + py * day[icell] + peta * daeta[icell]; //are these dax, day etc. the covariant components?

      //corrections to distribution function from shear stress
      double delta_f_shear = 0.0;
      if (INCLUDE_DELTAF)
      {
        double Wfactor = (ptau * ptau * pi00[icell] - 2.0 * ptau * px * pi01[icell] - 2.0 * ptau * py * pi02[icell] + px * px * pi11[icell] + 2.0 * px * py * pi12[icell] + py * py * pi22[icell] + peta * peta *pi33[icell]);
        delta_f_shear = ((1 - F0_IS_NOT_SMALL * sign_d[ipart] * f0) * Wfactor * deltaf_prefactor);
      }

      //corrections to distribution function from bulk pressure
      double delta_f_bulk = 0.0;
      if (INCLUDE_BULKDELTAF == 1)
      {
        if (bulk_deltaf_kind == 0) delta_f_bulk = (- (1. - F0_IS_NOT_SMALL * sign_d[ipart] * f0) * bulkPi_d[icell] * (bulkvisCoefficients[0] * mass_d[ipart] * mass_d[ipart] + bulkvisCoefficients[1] * pdotu + bulkvisCoefficients[2] * pdotu * pdotu));
        else if (bulk_deltaf_kind == 1)
        {
          double E_over_T = pdotu / Tdec_d[icell];
          double mass_over_T = mass_d[ipart] / Tdec_d[icell];
          delta_f_bulk = (-1.0 * (1. - sign_d[ipart] * f0)/E_over_T * bulkvisCoefficients[0] * (mass_over_T * mass_over_T / 3. - bulkvisCoefficients[1] * E_over_T * E_over_T) * bulkPi_d[icell]);
        }
        else if (bulk_deltaf_kind == 2)
        {
          double E_over_T = pdotu / Tdec_d[icell];
          delta_f_bulk = (-1. * (1. - sign_d[ipart] * f0) * (-bulkvisCoefficients[0] + bulkvisCoefficients[1] * E_over_T) * bulkPi_d[icell]);
        }
        else if (bulk_deltaf_kind == 3)
        {
          double E_over_T = pdotu / Tdec_d[icell];
          delta_f_bulk = (-1.0 * (1. - sign_d[ipart] * f0) / sqrt(E_over_T) * (-bulkvisCoefficients[0] + bulkvisCoefficients[1] * E_over_T) * bulkPi_d[icell]);
        }
        else if (bulk_deltaf_kind == 4)
        {
          double E_over_T = pdotu / Tdec_d[icell];
          delta_f_bulk = (-1.0 * (1. - sign_d[ipart] * f0) * (bulkvisCoefficients[0] - bulkvisCoefficients[1] / E_over_T) * bulkPi_d[icell]);
        }
      }

      double ratio = min(1., fabs(1. / (delta_f_shear + delta_f_bulk)));
      double result = prefactor * degen_d[ipart] * pdotdsigma * tau_d[icell] * f0 * (1. + (delta_f_shear + delta_f_bulk) * ratio);
      temp[icell] += result;

    }//finish if(icell < FO_length)
    int N = blockDim.x;
    __syncthreads(); //Make sure threads are prepared for reduction
    do
    {
      //Here N must be a power of two. Try reducing by powers of 2, 4, 6 etc...
      N /= 2;
      if (icell < N) temp[icell] += temp[icell + N];
      __syncthreads();//Test if this is needed
    } while(N != 1);

    long spectra_size = number_of_chosen_particles * pT_tab_length * phi_tab_length * y_tab_length;
    if (icell == 0) dN_pTdpTdphidy_d[blockIdx.x * spectra_size + imm] = temp[0];
	}
}

//Does a block sum, where the previous kernel did a thread sum.
__global__ void reduction(double* dN_pTdpTdphidy_d, int final_spectrum_size, int cooperfryeblocks)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < final_spectrum_size)
  {
    if (cooperfryeblocks == 1) return; //Probably will never happen, but best to be careful
    //Need to start at i=1, since adding everything to i=0
    for (int i = 1; i < cooperfryeblocks; i++) dN_pTdpTdphidy_d[idx] += dN_pTdpTdphidy_d[idx + i * final_spectrum_size];
  }
}

void EmissionFunctionArray::calculate_dN_ptdptdphidy_3DGPU()
{
  hipDeviceSynchronize();
  hipError_t err;

  long spectrum_size = ( (FO_length + threadsPerBlock - 1) / threadsPerBlock ) * number_of_chosen_particles * pT_tab_length * phi_tab_length * y_tab_length; //the size of the spectrum which has been intrablock reduced, but not interblock reduced
  int cooperfryeblocks = (FO_length + threadsPerBlock - 1) / threadsPerBlock; //?? number of blocks in the first kernel
  int final_spectrum_size = number_of_chosen_particles * pT_tab_length * phi_tab_length * y_tab_length; //size of final array for all particles , as a function of particle, pT and phi
  int blocks = (final_spectrum_size + threadsPerBlock -1) / threadsPerBlock; //?? number of blocks in the second kernel

  cout << "# of chosen particles   = " << number_of_chosen_particles << endl;
  cout << "FO_length               = " << FO_length      << endl;
  cout << "pT_tab_length           = " << pT_tab_length  << endl;
  cout << "phi_tab_length          = " << phi_tab_length << endl;
  cout << "y_tab_length          = " << y_tab_length << endl;
  cout << "unreduced spectrum size = " << spectrum_size  << endl; //?
  cout << "reduced spectrum size   = " << final_spectrum_size  << endl; //?
  cout << "threads per block       = " << threadsPerBlock<< endl;
  cout << "blocks in first kernel  = " << cooperfryeblocks<< endl;
  cout << "blocks in second kernel = " << blocks  << endl;

  //Convert object data into arrays to pass to the device
  //Particle Properties
  cout << endl << "Declaring host arrays" << endl;
  double  mass[number_of_chosen_particles];
  double  sign[number_of_chosen_particles];
  double  degen[number_of_chosen_particles];
  int     baryon[number_of_chosen_particles];

  //Momentum properties
  double  pT[pT_tab_length];
  double  trig[2*phi_tab_length]; //Contains sin then cos for discrete phi, no sense in calculating them 50,000 times

  //Freeze out surface properties
  double Tdec[FO_length];
  double Pdec[FO_length];
  double Edec[FO_length];

  //mu will enumerate mu for all particles before increasing to next cell
  //WHAT IS THIS?
  double *mu;
  mu = (double*) malloc( FO_length * number_of_chosen_particles * sizeof(double) );

  double Tdec[FO_length]
  double Pdec[FO_length]
  double Edec[FO_length]
  double mu[FO_length]
  double tau[FO_length]
  double eta[FO_length]
  double utau[FO_length]
  double ux[FO_length]
  double uy[FO_length]
  double ueta[FO_length]
  double datau[FO_length]
  double dax[FO_length]
  double day[FO_length]
  double daeta[FO_length]
  double pi00[FO_length]
  double pi01[FO_length]
  double pi02[FO_length]
  double pi11[FO_length]
  double pi12[FO_length]
  double pi22[FO_length]
  double pi33[FO_length]
  double muB[FO_length]
  double bulkPi[FO_length]

  cout << "declaring dN_pTdpTdphidy, hope it doesn't crash" << endl;
  double *dN_pTdpTdphidy;
  dN_pTdpTdphidy = (double*) malloc( spectrum_size * sizeof(double) );

  cout << "declared arrays" <<endl;

  //Fill arrays with data
  for(int i = 0; i < spectrum_size; i++) dN_pTdpTdphidy[i] = 0.0;
  for(int i = 0; i < number_of_chosen_particles; i++)
  {
    int particle_idx  = chosen_particles_sampling_table[i];
    particle_info *particle = &particles[particle_idx];

    mass[i]   = particle->mass  ;
    sign[i]   = particle->sign  ;
    degen[i]  = particle->gspin ;
    baryon[i] = particle->baryon;
  }
  for (int i = 0; i < pT_tab_length; i++) pT[i] = pT_tab->get(1, i+1);
  for (int i = 0; i < phi_tab_length; i++)
  {
    trig[i] = sin( phi_tab->get(1, i+1) );
    trig[i+phi_tab_length] = cos( phi_tab->get(1, i+1) );
  }
  for (int i = 0; i < y_tab_length; i++) y[i] = y_tab->get(1, i+1);

  for(int i = 0; i < FO_length; i++)
  {
    FO_surf *surf = &FOsurf_ptr[i];
    Tdec[i] = surf->Tdec;
    Pdec[i] = surf->Pdec;
    Edec[i] = surf->Edec;

    //WHAT IS THIS???
    for(int j = 0; j < number_of_chosen_particles; j++)
    {
      int particle_idx  = chosen_particles_sampling_table[j];
      mu[i * number_of_chosen_particles + j] = surf->particle_mu[particle_idx];
    }
    tau[i] = surf->tau;
    eta[i] = surf->eta;
    utau[i] = surf->u0;
    ux[i] = surf->u1;
    uy[i] = surf->u2;
    ueta[i] = surf->u3;
    datau[i] = surf->da0;
    dax[i] = surf->da1;
    day[i] = surf->da2;
    daeta[i] = surf->da3;
    pi00[i] = surf->pi00;
    pi01[i] = surf->pi01;
    pi02[i] = surf->pi02;
    pi11[i] = surf->pi11;
    pi12[i] = surf->pi12;
    pi22[i] = surf->pi22;
    pi33[i] = surf->pi33;
    muB[i] = surf->muB;
    bulkPi[i] = surf->bulkPi;
  }

  cout << "declaring device variables" << endl;
  //Make device copies of all of these arrays
  double *mass_d;
  double *sign_d;
  double *degen_d;
  int    *baryon_d;

  //Momentum properties
  double *pT_d;
  double *trig_d;
  double *hyperTrig_d;
  double *delta_eta_d;

  //Freeze out surface properties
  double *Tdec_d;
  double *Pdec_d;
  double *Edec_d;
  double *mu_d;
  double *tau_d;
  double *eta_d;
  double *utau_d;
  double *ux_d;
  double *uy_d;
  double *ueta_d;
  double *datau_d;
  double *dax_d;
  double *day_d;
  double *daeta_d;
  double *pi00_d;
  double *pi01_d;
  double *pi02_d;
  double *pi11_d;
  double *pi12_d;
  double *pi22_d;
  double *pi33_d;
  double *muB_d;
  double *bulkPi_d;

  double *dN_pTdpTdphidy_d;

  cout<< "allocating memory for device variables" << endl;
  //Allocate a lot of memory on device
  hipMalloc( (void**) &mass_d,   number_of_chosen_particles * sizeof(double)       );
  hipMalloc( (void**) &sign_d,   number_of_chosen_particles * sizeof(double)       );
  hipMalloc( (void**) &degen_d,  number_of_chosen_particles * sizeof(double)       );
  hipMalloc( (void**) &baryon_d, number_of_chosen_particles * sizeof(int)          );

  hipMalloc( (void**) &pT_d,     pT_tab_length * sizeof(double)                    );
  hipMalloc( (void**) &trig_d,   2 * phi_tab_length * sizeof(double)               );
  hipMalloc( (void**) &y_d,      y_tab_length * sizeof(double)                     );

  hipMalloc( (void**) &Tdec_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &Pdec_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &Edec_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &mu_d,     number_of_chosen_particles * FO_length * sizeof(double)  );
  hipMalloc( (void**) &tau_d,    FO_length * sizeof(double)                        );
  hipMalloc( (void**) &eta_d,    FO_length * sizeof(double)                        );
  hipMalloc( (void**) &utau_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &ux_d,     FO_length * sizeof(double)                        );
  hipMalloc( (void**) &uy_d,     FO_length * sizeof(double)                        );
  hipMalloc( (void**) &ueta_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &datau_d,  FO_length * sizeof(double)                        );
  hipMalloc( (void**) &dax_d,    FO_length * sizeof(double)                        );
  hipMalloc( (void**) &day_d,    FO_length * sizeof(double)                        );
  hipMalloc( (void**) &daeta_d,  FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi00_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi01_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi02_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi11_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi12_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi22_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &pi33_d,   FO_length * sizeof(double)                        );
  hipMalloc( (void**) &muB_d,    FO_length * sizeof(double)                        );
  hipMalloc( (void**) &bulkPi_d, FO_length * sizeof(double)                        );

  hipMalloc( (void**) &dN_pTdpTdphidy_d, spectrum_size * sizeof(double)            );

  cout << "Finished allocating device memory" << endl;
  cout << "Copying data from host to device" << endl;

  //Copy the CPU variables to GPU
  hipMemcpy( mass_d,   	mass,   number_of_chosen_particles * sizeof(double),   hipMemcpyHostToDevice );
  hipMemcpy( sign_d,   	sign,   number_of_chosen_particles * sizeof(double),   hipMemcpyHostToDevice );
  hipMemcpy( degen_d,  	degen,  number_of_chosen_particles * sizeof(double),   hipMemcpyHostToDevice );
  hipMemcpy( baryon_d, 	baryon, number_of_chosen_particles * sizeof(int),      hipMemcpyHostToDevice );
  hipMemcpy( pT_d,     	pT,     pT_tab_length * sizeof(double),                hipMemcpyHostToDevice );
  hipMemcpy( trig_d,   	trig,   2*phi_tab_length * sizeof(double),             hipMemcpyHostToDevice );
  hipMemcpy( Tdec_d,   	Tdec,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( Pdec_d,   	Pdec,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( Edec_d,   	Edec,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( mu_d,     	mu,     number_of_chosen_particles*FO_length*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy( tau_d,    	tau,    FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( eta_d,    	eta,    FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( utau_d, 	  utau,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( ux_d,     	ux,     FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( uy_d,     	uy,     FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( ueta_d,     ueta,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( datau_d,    datau,  FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( dax_d,    	dax,    FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( day_d,    	day,    FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( daeta_d,    daeta,  FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi00_d,   	pi00,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi01_d,   	pi01,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi02_d,   	pi02,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi11_d,   	pi11,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi12_d,   	pi12,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi22_d,   	pi22,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( pi33_d,   	pi33,   FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( muB_d,   	  muB,    FO_length * sizeof(double),                    hipMemcpyHostToDevice );
  hipMemcpy( bulkPi_d,   bulkPi, FO_length * sizeof(double),                    hipMemcpyHostToDevice );

  hipMemcpy( dN_pTdpTdphidy_d, dN_pTdpTdphidy, spectrum_size * sizeof(double),  hipMemcpyHostToDevice ); //this is an empty array, so why do we need to memcpy it?

  cout << "Finished copying from host to device." << endl;

  //Perform kernels, first inital cooper Frye and reduction acriss threads, second is another reduction across blocks
  double prefactor = 1.0 / (8.0 * (M_PI * M_PI * M_PI)) / hbarC / hbarC / hbarC;

  if(debug) cout << "Starting first cooper-frye kernel" << endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  hipDeviceSynchronize();

  cooperFrye3D<<<cooperfryeblocks, threadsPerBlock>>>(FO_length, number_of_chosen_particles, pT_tab_length, phi_tab_length, y_tab_length,
                                dN_pTdpTdphidy_d, pT_d, trig_d, y_d,
                                mass_d, sign_d, degen_d, baryon_d,
                                Tdec_d, Pdec_d, Edec_d, mu_d, tau_d, eta_d,
                                utau_d, ux_d, uy_d, ueta_d,
                                datau_d, dax_d, day_d, daeta_d,
                                pi00_d, pi01_d, pi02_d, pi11_d, pi12_d, pi22_d, pi33_d,
                                muB_d, bulkPi_d,
                                hbarC, bulk_deltaf_kind, INCLUDE_DELTAF, INCLUDE_BULKDELTAF, F0_IS_NOT_SMALL)
  hipDeviceSynchronize();

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("Error in first kernel: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  cout << "Finished first kernel" << endl;

  cout << "Starting second kernel" << endl;

  hipDeviceSynchronize();
  reduction<<<blocks, threadsPerBlock>>>(dN_pTdpTdphidy_d, final_spectrum_size, cooperfryeblocks);
  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  float seconds = milliseconds * 1000.0;

  cout << "Finished in " << seconds << " seconds." << endl;

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("Error in second kernel: %s\n", hipGetErrorString(err));
    err = hipSuccess;
  }

  //cout << "finished second kernel" << endl;

  //Copy spectra back to host
  cout << "Copyying spectra from device to host" << endl;

  hipMemcpy( dN_pTdpTdphidy, dN_pTdpTdphidy_d, number_of_chosen_particles * pT_tab_length * phi_tab_length * y_tab_length * sizeof(double),  hipMemcpyDeviceToHost );

  cout << "Finished copying data back to host" << endl;
  cout << "Writing spectra to files" << endl;

  //Write results to files
  ofstream of1(dN_ptdptdphidy_filename.c_str(), ios_base::app);

  //NEED A NEW CONVENTION/FORMAT FOR SPECTRA FILES...
  for(int i=0; i<Nparticles; i++)
  {
    //The bouncer. See if a particle is chosen.
    for(int counter = 0; counter< number_of_chosen_particles; counter++)
    {
      if(chosen_particles_sampling_table[counter] == i)
      {
        for (int phiIdx=0; phiIdx<phi_tab_length; phiIdx++)
        {
          for (int pTIdx=0; pTIdx<pT_tab_length; pTIdx++)
          {
            of1 << scientific <<  setw(15) << setprecision(8) << dN_pTdpTdphidy[pTIdx + pT_tab_length*phiIdx + counter*pT_tab_length*phi_tab_length] << "  ";
          }
          of1 << endl; //Only new lines for new angles
        }
        break;
      }
      //If counter reaches max value, particle wasnt chosen, output zeros
      if(counter == number_of_chosen_particles - 1)
      {
        for (int phiIdx=0; phiIdx<phi_tab_length; phiIdx++)
        {
          for (int pTIdx=0; pTIdx<pT_tab_length; pTIdx++)
          {
            of1 << scientific <<  setw(15) << setprecision(8) << 0.  << "  ";
          }
          of1 << endl; //Only new lines for new angles
        }
      }
    }
  }
  of1.close();


  cout << "finished writing spectra to files" << endl;
  cout << "Freeing device memory and cpu memory" << endl;

  //Free Memory
  free( mu );
  free( dN_pTdpTdphidy );
  hipFree( mass_d );
  hipFree( sign_d );
  hipFree( degen_d );
  hipFree( baryon_d );
  hipFree( pT_d );
  hipFree( trig_d );
  hipFree( hyperTrig_d );
  hipFree( delta_eta_d );
  hipFree( Tdec_d );
  hipFree( Pdec_d );
  hipFree( Edec_d );
  hipFree( mu_d );
  hipFree( tau_d );
  hipFree( gammaT_d );
  hipFree( ux_d );
  hipFree( uy_d );
  hipFree( da0_d );
  hipFree( da1_d );
  hipFree( da2_d );
  hipFree( pi00_d );
  hipFree( pi01_d );
  hipFree( pi02_d );
  hipFree( pi11_d );
  hipFree( pi12_d );
  hipFree( pi22_d );
  hipFree( pi33_d );
  hipFree( muB_d  );
  hipFree( bulkPi_d );
  hipFree( dN_pTdpTdphidy_d );
  cout << "Finished everything! Have a good day!" << endl;
}


#endif
